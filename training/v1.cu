#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>

using namespace std;

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

__global__ void mykernel(float* r, const float* d, int n) {
    // TO DO
}

static inline int divup(int a, int b) {
    return (a + b - 1) / b;
}

void step(float* r, const float* d, int n) {
    // Allocate memory & copy data to GPU
    float* dGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, n * n * sizeof(float)));
    float* rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(n, dimBlock.x), divup(n, dimBlock.y));
    mykernel << <dimGrid, dimBlock >> > (rGPU, dGPU, n);
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}

int main() {
    constexpr int n = 20000;
    // Generate a random graph
    vector<float> d(n * n);
    srand(static_cast<unsigned>(time(nullptr)));
    for (int i = 0; i < n * n; ++i) {
        d[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    // Compute shortest 2-edge paths
    vector<float> r(n * n);
    auto start = chrono::high_resolution_clock::now();
    step(r.data(), d.data(), n);
    auto end = chrono::high_resolution_clock::now();
    // // Display results
    // for (int i = 0; i < n; ++i) {
    //     for (int j = 0; j < n; ++j) {
    //         cout << r[i * n + j] << " ";
    //     }
    //     cout << "\n";
    // }
    chrono::duration<float> duration = end - start;
    cout << "Time elapsed: " << duration.count() << " seconds\n";
    return 0;
}